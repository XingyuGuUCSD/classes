#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;


#ifndef TYPE 
#define TYPE 6
#endif

#if TYPE >= 4
#define BLOCK_SIZE 16
#else
#define BLOCK_SIZE 32
#endif

#define ASUB(i, j) Asub[i][j]
#define BSUB(i, j) Bsub[i][j]

//NOTE: B MUST NOT be transposed
__global__ void matMulNaive(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y; // Row i of C
    int J =  blockIdx.x*blockDim.x + threadIdx.x; // Row J of C

    if((I < N) && (J < N)){
        _DOUBLE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _DOUBLE_ a = A[I * N + k];
            _DOUBLE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}

//NOTE: B must NOT be transposed
__global__ void matMulTiled(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {
    
    // Block && thread index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // shared partial memory for subPartialroblems of A & B
    __shared__ _DOUBLE_ Asub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ _DOUBLE_ Bsub[BLOCK_SIZE][BLOCK_SIZE];

    // temp variable
    _DOUBLE_ partialC = 0;

    // pull out termination
    int terminate = (N * BLOCK_SIZE * by) + N - 1;
    int b = BLOCK_SIZE * bx;
    int bInc = BLOCK_SIZE * N;

    // Loop over all the subblocks
    for (int a = N * BLOCK_SIZE * by; a <= terminate;a += BLOCK_SIZE, b += bInc) {

        // Loading the arrays from memory
        ASUB(ty, tx) = A[a + N * ty + tx];
        BSUB(tx, ty) = B[b + N * tx + ty];
        __syncthreads();

        // Perform the multiply
        for (int k = 0; k < BLOCK_SIZE; ++k)
            partialC += ASUB(ty, k) * BSUB(k,tx);
        
        // Wait for all to finish before moving to next iteration 
        __syncthreads();
    }

    // Finally write out the result in c to C in memory
    C[(N * BLOCK_SIZE * by + BLOCK_SIZE * bx) + N * ty + tx] = partialC;
}

//Note: B MUST be transposed
__global__ void matMulCoalesced(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    // Block && thread index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // shared partial memory for subPartialroblems of A & B
    __shared__ _DOUBLE_ Asub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ _DOUBLE_ Bsub[BLOCK_SIZE][BLOCK_SIZE];

    // temp variable
    _DOUBLE_ partialC = 0;

    // pull out termination
    int terminate = (N * BLOCK_SIZE * by) + N - 1;
    int b = BLOCK_SIZE * bx;
    int bInc = BLOCK_SIZE * N;

    // Loop over all the subblocks
    for (int a = N * BLOCK_SIZE * by; a <= terminate; a += BLOCK_SIZE, b += bInc) {

        // Loading the arrays from memory
        ASUB(ty, tx) = A[a + N * ty + tx];
        BSUB(tx, ty) = B[b + N * ty + tx];
        __syncthreads();
	
	// Perform the multiply
        for (int k = 0; k < BLOCK_SIZE; ++k)
            partialC += ASUB(ty, k) * BSUB(tx,k);

        // Wait for all to finish before moving to next iteration 
        __syncthreads();
    }

    // Finally write out the result in c to C in memory
    C[( N * BLOCK_SIZE * by + BLOCK_SIZE * bx) + N * ty + tx] = partialC;
}

//NOTE: B MUST be transposed
__global__ void matMulBankConflict(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

    // Block && thread index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // shared partial memory for subPartialroblems of A & B
    __shared__ _DOUBLE_ Asub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ _DOUBLE_ Bsub[BLOCK_SIZE][BLOCK_SIZE];

    // temp variable
    _DOUBLE_ partialC = 0;

    // pull out termination
    int terminate = (N * BLOCK_SIZE * by) + N - 1;
    int b = BLOCK_SIZE * bx;
    int bInc = BLOCK_SIZE * N;

    // Loop over all the subblocks
    for (int a = N * BLOCK_SIZE * by; a <= terminate; a += BLOCK_SIZE, b += bInc){

        // Loading the arrays from memory
        ASUB(ty, tx) = A[a + N * ty + tx];
        BSUB(ty, tx) = B[b + N * ty + tx];
        __syncthreads();

        // Perform the multiply
	for (int k = 0; k < BLOCK_SIZE; ++k)
            partialC += ASUB(ty, k) * BSUB(k,tx);
        
        // Wait for all to finish before moving to next iteration 
        __syncthreads();

    }

    // Finally write out the result in c to C in memory
    C[(N * BLOCK_SIZE * by + BLOCK_SIZE * bx) + N * ty + tx] = partialC;
}

//NOTE: B MUST be transposed
__global__ void
matMulOuterProduct(int N, _DOUBLE_* C, _DOUBLE_* A, _DOUBLE_* B)
{
    // Block && thread index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // shared partial memory for subPartialroblems of A
    __shared__ _DOUBLE_ Asub[BLOCK_SIZE * BLOCK_SIZE];

    // create a temporary array to store partial C
    _DOUBLE_ cArr[BLOCK_SIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    
    // pull out termination
    int terminate = (N * BLOCK_SIZE * by) + N -1;
    int b = BLOCK_SIZE * 4 * bx;
    int bInc  = BLOCK_SIZE * N;

    // loop of all sub blocks
    for (int a = N * BLOCK_SIZE * by; a <= terminate; a += BLOCK_SIZE, b += bInc) {
      // load the sub block of A
      for(int i = 0; i < 4; i++){
        Asub[ (i*4+ty) + BLOCK_SIZE * tx] = A[a + N * (i*4+ty) + tx];
      }
      __syncthreads();

      _DOUBLE_ *aPartial = &Asub[0];
      _DOUBLE_ *bPartial = &B[b + BLOCK_SIZE * ty + tx];
     
      // compute the partial sums
      for(int i = 0; i < BLOCK_SIZE; i++){
        _DOUBLE_ bVal = bPartial[0];
        cArr[0] +=  aPartial[0] * bVal;
        cArr[1] +=  aPartial[1] * bVal;
        cArr[2] +=  aPartial[2] * bVal;
        cArr[3] +=  aPartial[3] * bVal;
        cArr[4] +=  aPartial[4] * bVal;
        cArr[5] +=  aPartial[5] * bVal;
        cArr[6] +=  aPartial[6] * bVal;
        cArr[7] +=  aPartial[7] * bVal;
        cArr[8] +=  aPartial[8] * bVal;
        cArr[9] +=  aPartial[9] * bVal;
        cArr[10] +=  aPartial[10] * bVal;
        cArr[11] +=  aPartial[11] * bVal;
        cArr[12] +=  aPartial[12] * bVal;
        cArr[13] +=  aPartial[13] * bVal;
        cArr[14] +=  aPartial[14] * bVal;
        cArr[15] +=  aPartial[15] * bVal;
        aPartial += BLOCK_SIZE;
        bPartial += N;
      }

      // sync all threads
      __syncthreads();
    }

    // store back to memory
    for(int i = 0; i < BLOCK_SIZE; i++){
      C[((N * BLOCK_SIZE * by + 4 * BLOCK_SIZE * bx) + BLOCK_SIZE * ty + tx) + N * i] = cArr[i];
    }

}

//NOTE: B MUST be transposed
__global__ void
matMulPrefetch( int N, _DOUBLE_* C, _DOUBLE_* A, _DOUBLE_* B)
{

    // Block && thread index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // shared partial memory for subPartialroblems of A & B
    __shared__ _DOUBLE_ Asub[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ _DOUBLE_ Asub2[BLOCK_SIZE * BLOCK_SIZE];

    _DOUBLE_ *prefetch = Asub;
    _DOUBLE_ *prefetch2 = Asub2;

    _DOUBLE_ cArr[BLOCK_SIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};


    // pull out terminate
    int terminate = (N * BLOCK_SIZE * by) + N -1;
    int b = BLOCK_SIZE * 4 * bx;
    int bInc  = BLOCK_SIZE * N;
    int cStart = N * BLOCK_SIZE * by + 4 * BLOCK_SIZE * bx;

    //prefetch A
      _DOUBLE_ *Aprefetch = &A[(N * BLOCK_SIZE * by) + N * ty +tx];
      _DOUBLE_ *aPartial = &prefetch[ty + BLOCK_SIZE * tx];
      
      for(int i = 0; i < 16; i+=4){
        aPartial[i] = Aprefetch[N * i];
      }
      __syncthreads();

    // loop of all sub blocks
    for (int a = N * BLOCK_SIZE * by; a <= terminate; a += BLOCK_SIZE, b += bInc) {
      // load the sub block of A
      Aprefetch = &A[a + BLOCK_SIZE + N * ty +tx];
      _DOUBLE_ *aPartial2 = &prefetch2[ty + BLOCK_SIZE * tx];

      for(int i = 0; i < 16; i+=4){
        aPartial2[i] = Aprefetch[N * i];
      }

      aPartial = &prefetch[0];
      _DOUBLE_ *bPartial = &B[b + BLOCK_SIZE * ty + tx];

      for(int i = 0; i < BLOCK_SIZE; i++){
        _DOUBLE_ bVal = bPartial[0];
        cArr[0] +=  aPartial[0] * bVal;
        cArr[1] +=  aPartial[1] * bVal;
        cArr[2] +=  aPartial[2] * bVal;
        cArr[3] +=  aPartial[3] * bVal;
        cArr[4] +=  aPartial[4] * bVal;
        cArr[5] +=  aPartial[5] * bVal;
        cArr[6] +=  aPartial[6] * bVal;
        cArr[7] +=  aPartial[7] * bVal;
        cArr[8] +=  aPartial[8] * bVal;
        cArr[9] +=  aPartial[9] * bVal;
        cArr[10] +=  aPartial[10] * bVal;
        cArr[11] +=  aPartial[11] * bVal;
        cArr[12] +=  aPartial[12] * bVal;
        cArr[13] +=  aPartial[13] * bVal;
        cArr[14] +=  aPartial[14] * bVal;
        cArr[15] +=  aPartial[15] * bVal;
        aPartial += BLOCK_SIZE;
        bPartial += N;
      }

      // Synchronize to make sure the matrices are loaded
      __syncthreads();

      // swaPartial Asub and Asub2
      _DOUBLE_ *prefetch_temp = prefetch;
      prefetch = prefetch2;
      prefetch2 = prefetch_temp;

    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    _DOUBLE_ *Cpartial = &C[cStart];
    Cpartial += BLOCK_SIZE * ty + tx;
    for(int i=0; i<BLOCK_SIZE; i++){
      Cpartial[0] = cArr[i]; 
      Cpartial += N;
    }

}

//NOTE: B MUST BE transposed
__global__ void
matMulUnroll( int N, _DOUBLE_* C, _DOUBLE_* A, _DOUBLE_* B)
{
    // Block && thread index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // shared partial memory for subPartialroblems of A & B
    __shared__ _DOUBLE_ Asub[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ _DOUBLE_ Asub2[BLOCK_SIZE * BLOCK_SIZE];

    _DOUBLE_ *prefetch = Asub;
    _DOUBLE_ *prefetch2 = Asub2;

    _DOUBLE_ cArr[BLOCK_SIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

    // pull out terminate
    int terminate = (N * BLOCK_SIZE * by) + N - 1;
    int b = BLOCK_SIZE * 4 * bx;
    int bInc  = BLOCK_SIZE * N;
    int cStart = N * BLOCK_SIZE * by + 4 * BLOCK_SIZE * bx;

    // prefetch
      _DOUBLE_ *Aprefetch = &A[(N * BLOCK_SIZE * by) + N * ty +tx];
      _DOUBLE_ *aPartial = &prefetch[ty + BLOCK_SIZE * tx];
#pragma unroll
      for(int i = 0; i < 16; i+=4){
        aPartial[i] = Aprefetch[N * i];
      }
      __syncthreads();

    // loop of all sub blocks
    for (int a = N * BLOCK_SIZE * by; a <= terminate; a += BLOCK_SIZE, b += bInc) {

      Aprefetch = &A[a + BLOCK_SIZE + N * ty +tx];
      _DOUBLE_ *aPartial2 = &prefetch2[ty + BLOCK_SIZE * tx];
#pragma unroll
      for(int i = 0; i < 16; i+=4){
        aPartial2[i] = Aprefetch[N * i];
      }

      aPartial = &prefetch[0];
      _DOUBLE_ *bPartial = &B[b + BLOCK_SIZE * ty + tx];

#pragma unroll
      for(int i = 0; i < BLOCK_SIZE; i++){
        _DOUBLE_ bVal = bPartial[0];
        cArr[0] +=  aPartial[0] * bVal;
        cArr[1] +=  aPartial[1] * bVal;
        cArr[2] +=  aPartial[2] * bVal;
        cArr[3] +=  aPartial[3] * bVal;
        cArr[4] +=  aPartial[4] * bVal;
        cArr[5] +=  aPartial[5] * bVal;
        cArr[6] +=  aPartial[6] * bVal;
        cArr[7] +=  aPartial[7] * bVal;
        cArr[8] +=  aPartial[8] * bVal;
        cArr[9] +=  aPartial[9] * bVal;
        cArr[10] +=  aPartial[10] * bVal;
        cArr[11] +=  aPartial[11] * bVal;
        cArr[12] +=  aPartial[12] * bVal;
        cArr[13] +=  aPartial[13] * bVal;
        cArr[14] +=  aPartial[14] * bVal;
        cArr[15] +=  aPartial[15] * bVal;
        aPartial += BLOCK_SIZE;
        bPartial += N;
      }

      __syncthreads();

      // swap Asub and Asub2
      _DOUBLE_ *prefetch_temp = prefetch;
      prefetch = prefetch2;
      prefetch2 = prefetch_temp;

    }

    //write back to memory
    _DOUBLE_ *Cpartial = &C[cStart];
    Cpartial += BLOCK_SIZE * ty + tx;
#pragma unroll
    for(int i=0; i<BLOCK_SIZE; i++){
      Cpartial[0] = cArr[i]; 
      Cpartial += N;
    }

}
