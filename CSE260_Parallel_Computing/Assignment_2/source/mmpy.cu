/*
 * Simplest matrix multiplication in CUDA
 *
 * Scott B. Baden, University of California, San Diego
 * April 2010
 *
 * We compute C = A * B
 *
 * This code assumes that the  matrices are square though there
 * are hooks to facilitate  extending the code to non-square matrices
 *
 */

// system includes
#include <stdio.h>
#include <assert.h>

//  include the kernel
#include "mmpy_kernel.cu"

#include "types.h"
#include "utils.h"

#ifndef TYPE
#define TYPE 6
#endif

// External function definitions
void genMatrix( _DOUBLE_ *a, unsigned int m, unsigned int n);
void genMatrixTranspose( _DOUBLE_ *a, unsigned int m, unsigned int n);
void verify( _DOUBLE_ *c, unsigned int m, unsigned int n, _DOUBLE_ eps, char *mesg);
void verify( _DOUBLE_ *c_d, _DOUBLE_ *c_h,  unsigned int m, unsigned int n, _DOUBLE_ eps, char *mesg);
void printMatrix( _DOUBLE_ *a, unsigned int m, unsigned int n);
void cmdLine(int argc, char *argv[], int& n, int& reps, int& ntx, int& nty, _DOUBLE_ & eps, int& do_host, int& prefer_l1);
void perfString(int n, int ntx, int nty, int reps, double t_h, double gflops_h, double t_d, double gflops_d);
// extern "C"{
    double getTime();
    double gflops(int n, int niter, double time);
//}
void matMulHost(_DOUBLE_ *, const _DOUBLE_ *, const _DOUBLE_ *, unsigned int, unsigned int);
void setGrid(int n, dim3 &blockDim, dim3 &gridDim);

int
main(int argc, char** argv) {
    // To improve repeatabilty of measurements taken on the device,
    // we multiply the number of reps by this scale factor
    // Adjust as needed
    const int SCALE = 10;

// Read in the command line elements
    int n, reps, ntx, nty, do_host, prefer_l1;
    _DOUBLE_ eps;

    cmdLine(argc, argv, n, reps, ntx, nty, eps, do_host, prefer_l1);

   // The thread geometry must evenly divide N
   /*if ((n % ntx != 0) || (n % nty != 0) )
   {
        printf("Thread geometry: %d x %d\n",ntx, nty);
        printf("The length of the thread geometry axis ");
        printf("[ %d x %d]\n",ntx, nty);
        printf("  nust divide N [%d] evenly\n",n);
        exit(-1);
   }
   */

    // Total amount of storage for entries
    unsigned int n2 = n*n*sizeof(_DOUBLE_);

    // Select the fastest device and report characteristics
    int major, minor;
    selectAndReport(&major, &minor);
#ifdef _DOUBLE
    if ((major == 1) && (minor == 2)){
        printf("   You are running on a capability 1.2 device.\n");
        printf("   This code has been compiled with double precision arithmetic.\n");
	printf("   Recompile with single precision.\n\n");
	exit(-1);
    }
#endif

    // setup execution configurations
    int _ntx, _nty;
#if TYPE >= 4
    //_ntx = ntx;
    //_nty = nty;
    _ntx = 16;
    _nty = 4;
#else
    _ntx = 32;
    _nty = 32;
#endif

    dim3 threads(_ntx, _nty,1);
    int numblocksX = n/_ntx;
    int numblocksY = n/_nty;

    if( n % _ntx != 0  )
        numblocksX++;

    if( n % _nty != 0  )
        numblocksY++;

#if TYPE >= 4
    numblocksX = n/64;
    numblocksY = n/16;
    dim3 grid(numblocksX, numblocksY, 1);
#else
    dim3 grid(numblocksX, numblocksY, 1);
    //setGrid(n, threads, grid);
#endif

    // print configurations
    printf("n: %d, tx: %d, ty: %d, gridX: %d, gridY: %d, reps: %d, epsilon: %g\n\n", n, threads.x, threads.y, grid.x, grid.y, reps, eps);

  
#ifndef _DOUBLE
    printf("Using Single precision arithmetic\n\n");
#else
    printf("Using Double precision arithmetic\n\n");
#endif

#if (TYPE == 0)
    printf("Using naive GPU algorithm\n\n");
#elif (TYPE == 1) 
    printf("Using Tiled GPU algorithm\n\n");
#elif (TYPE == 2) 
    printf("Using +Coalesced GPU algorithm\n\n");
#elif (TYPE == 3) 
    printf("Using +NoBankComflicts GPU algorithm\n\n");
#elif (TYPE == 4)
    printf("Using +outerProduct GPU algorithm\n\n");
#elif (TYPE == 5)
    printf("Using +prefetch GPU algorithm\n\n");
#elif (TYPE == 6) 
    printf("Using +unroll GPU algorithm\n\n");
#endif

    if (do_host)
        printf("Doing host computation for comparison\n");

    // allocate an initialize host memory for A and B matrices
    _DOUBLE_ *h_A = (_DOUBLE_ *) malloc(n2);
    assert(h_A);
    _DOUBLE_ *h_B = (_DOUBLE_ *) malloc(n2);
    assert(h_B);
    genMatrix(h_A, n, n);
#if (TYPE == 0 || TYPE == 1)
    genMatrix(h_B, n, n);
#else
    genMatrixTranspose(h_B, n, n);
#endif

    if (n <= 8){
        printf("\nA:\n");
        printMatrix( h_A, n,n);
        printf("\nB:\n");
        printMatrix( h_B, n,n);
    }

    _DOUBLE_  *hostC;
    double t_host=0.0, gflops_h=0.0;
    if (do_host){
        // compute matrix product on the host
        hostC = (_DOUBLE_ *) malloc(n2);
        t_host = -getTime();
        for (int r=0; r< reps; r++)
            matMulHost(hostC, h_A, h_B, n, n);
        t_host += getTime();
        gflops_h = gflops(n, reps, t_host );
        printf("Host computation time: %f sec. [%f gflops]\n",t_host,gflops_h);

        // Verify host result
        verify( hostC,n,n,eps, "Host result");

        if (n <= 8){
            printf("\nC:\n");
            printMatrix( hostC, n,n);
        }
    }

    // allocate device memory
    _DOUBLE_ *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, n2);
    checkCUDAError("Error allocating device memory for matrix A");
    hipMalloc((void**) &d_B, n2);
    checkCUDAError("Error allocating device memory for matrix B");
    hipMalloc((void**) &d_C, n2);
    checkCUDAError("Error allocating device memory for matrix C");
    hipMemset((void **) d_A,-99,n2);
    checkCUDAError("Error initializing device memory matrix A");
    hipMemset((void **) d_B,-99,n2);
    checkCUDAError("Error initializing device memory matrix B");
    hipMemset((void **) d_C,0,n2);
    checkCUDAError("Error clearing device memory matrix C");

    // copy host memory to device
    hipMemcpy(d_A, h_A, n2, hipMemcpyHostToDevice);
    checkCUDAError("Error copying matrix A to device");
    hipMemcpy(d_B, h_B, n2, hipMemcpyHostToDevice);
    checkCUDAError("Error copying matrix B to device");


    // allocate host memory for the result
    _DOUBLE_  *h_C = (_DOUBLE_ *) malloc(n2);
    assert(h_C);


// If we set the preference for L1 cache, rather than
// shared memory, we may run slightly faster on devices that have the capability
    hipFuncCache_t Preference;
    if (prefer_l1){
        Preference = hipFuncCachePreferL1;
    }
    else{
        Preference = hipFuncCachePreferShared;
    }
#if (TYPE == 0)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulNaive),Preference);
#elif (TYPE == 1)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulTiled),Preference);
#elif (TYPE == 2)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulCoalesced),Preference);
#elif (TYPE == 3)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulBankConflict),Preference);
#elif (TYPE == 4)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulOuterProduct),Preference);
#elif (TYPE == 5)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulPrefetch),Preference);
#elif (TYPE == 6)
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMulUnroll),Preference);
#endif


    // Start the timer
#ifdef CUDA_TIMER
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event) ;
    hipEventCreate(&stop_event);
#endif

#ifdef CUDA_TIMER
    hipEventRecord(start_event, 0);
    float t_device;
#else
    hipDeviceSynchronize();
    double t_device = -getTime();
#endif

    // execute the kernel
    for (int r=0; r< SCALE*reps; r++) {
#if (TYPE == 0)        
	matMulNaive<<< grid, threads >>>(n, d_C, d_A, d_B);
#elif (TYPE == 1)
        matMulTiled<<< grid, threads >>>(n, d_C, d_A, d_B);
#elif (TYPE == 2)
        matMulCoalesced<<< grid, threads >>>(n, d_C, d_A, d_B);
#elif (TYPE == 3)
        matMulBankConflict<<< grid, threads >>>(n, d_C, d_A, d_B);
#elif (TYPE == 4)
        matMulOuterProduct<<< grid, threads >>>(n, d_C, d_A, d_B);
#elif (TYPE == 5)
        matMulPrefetch<<< grid, threads >>>(n, d_C, d_A, d_B);
#elif (TYPE == 6)
        matMulUnroll<<< grid, threads >>>(n, d_C, d_A, d_B);
#endif
}

#ifdef CUDA_TIMER
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&t_device, start_event, stop_event);
    t_device /= 1000.0;

#else
    // block until the device has finished
    hipDeviceSynchronize();
    // Stop the timer
    t_device +=getTime();
#endif

    checkCUDAError("Error in matrixMul kernel");

    // copy result from device to host
    hipMemcpy(h_C, d_C, n2, hipMemcpyDeviceToHost);
    checkCUDAError("Unable to retrieve result from device");



    double gflops_d = gflops(n, SCALE*reps, t_device );
    printf("Device computation time: %f sec. [%f gflops]\n",t_device,gflops_d);
    perfString(n, ntx, nty, reps, t_host, gflops_h, t_device, gflops_d);

    // Verify the device result
    verify( h_C,n,n,eps, "Device result");

    if (do_host)
        // Compare host and device results
        verify( h_C, hostC, n, n,eps,"Device vs. host");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    if (do_host)
        free(hostC);

    assert(hipSuccess ==hipFree(d_A));
    assert(hipSuccess ==hipFree(d_B));
    assert(hipSuccess ==hipFree(d_C));

    hipDeviceReset();
}
